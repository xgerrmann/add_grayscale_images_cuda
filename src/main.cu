#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

__global__ void combine(uchar *a, uchar *b, uchar *c){
	c[blockIdx.x] = (a[blockIdx.x] + b[blockIdx.x])/2;
}

int main(){
	// Load images
	const char *im0path = "media/im0.png";
	const char *im1path = "media/im1.png";
	cv::Mat im0_tmp = cv::imread(im0path,0);
	cv::Mat im1_tmp = cv::imread(im1path,0);
	uchar *im0, *im1, *im2;
	uchar *im0_c, *im1_c, *im2_c;
	int N = im0_tmp.rows*im0_tmp.cols; // Number of pixels
	std::cerr << "# Pixels: " << N << std::endl;
	int size = N * sizeof(uchar);
	
	// Show original images
	cv::imshow(im0path,im0_tmp);
	cv::imshow(im1path,im1_tmp);
	cv::waitKey(0);

	// Allocate space for device copies
	hipMalloc((void **)&im0_c,size);
	hipMalloc((void **)&im1_c,size);
	hipMalloc((void **)&im2_c,size);

	// Allocate space for host copies
	im0 = im0_tmp.ptr();
	im1 = im1_tmp.ptr();
	im2 = (uchar *)malloc(size);

	// Copy inputs to device
	hipMemcpy(im0_c, im0, size, hipMemcpyHostToDevice);
	hipMemcpy(im1_c, im1, size, hipMemcpyHostToDevice);
	
	// Execute combine on GPU
	combine<<<N,1>>>(im0_c, im1_c, im2_c); // block indexing
	
	// Copy result to host
	hipMemcpy(im2, im2_c, size, hipMemcpyDeviceToHost);

	// Cleanup
	cv::Mat im_out(im0_tmp.rows, im0_tmp.cols, im0_tmp.type(), im2);
	cv::imshow("im_out",im_out);
	cv::waitKey(0);
	
	free(im2); // im0 and im1 memory is managed by opencv.
	hipFree(im0_c); hipFree(im1_c);

	return 0;
}
